#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "./kernel.cuh"



__global__ void update_board_kernel(const unsigned char* cells, unsigned char* new_cells, const unsigned int boardSize)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < boardSize && j < boardSize)
    {
		int count = 0;
        for (int di = -1; di <= 1; ++di)
        {
            for (int dj = -1; dj <= 1; ++dj)
            {
				if (di == 0 && dj == 0)
					continue;

				int ni = i + di;
				int nj = j + dj;

                if (ni >= 0 && ni < boardSize && nj >= 0 && nj < boardSize && cells[ni * boardSize + nj] == 1)
                {
					count++;
				}
			}
		}

        if (cells[i * boardSize + j] == 1)
        {
            if (count < 2 || count > 3)
            {
				new_cells[i * boardSize + j] = 0;
			}
            else
            {
				new_cells[i * boardSize + j] = 1;
			}
		}
        else
        {
            if (count == 3)
            {
				new_cells[i * boardSize + j] = 1;
			}
            else
            {
				new_cells[i * boardSize + j] = 0;
			}
		}
	}
}

void update_board(const unsigned char* cells, unsigned char* new_cells, const unsigned int boardSize)
{
    unsigned char* d_cells, * d_new_cells;
    hipMalloc((void**)&d_cells, boardSize * boardSize * sizeof(unsigned char));
    hipMalloc((void**)&d_new_cells, boardSize * boardSize * sizeof(unsigned char));

    hipMemcpy(d_cells, cells, boardSize * boardSize * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_new_cells, new_cells, boardSize * boardSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize(boardSize / 16 + 1, boardSize / 16 + 1);

    update_board_kernel << <gridSize, blockSize >> > (d_cells, d_new_cells, boardSize);

    hipMemcpy(new_cells, d_new_cells, boardSize * boardSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_cells);
    hipFree(d_new_cells);
}