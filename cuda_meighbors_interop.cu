#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "./kernel.cuh"

#include <stdio.h>
#include <cassert>

unsigned char* interop_cells;
unsigned char* interop_cells2;
unsigned char* interop_uploadCells;
unsigned char* interop_neighbors;
char* interop_dirty;
unsigned int interop_boardSize;

const int numCellsPerIteration = 64;
const int sliceSize = 9;

__global__ void interop_count_intitial_neighbors(unsigned char* cells, unsigned char* neighbors, unsigned int boardSize)
{
    for (unsigned int i = 0; i < boardSize; i++)
    {
        for (unsigned int j = 0; j < boardSize; j++)
        {
            int index = i * boardSize + j;
            int numNeighbors = 0;
            for (int i2 = -1; i2 <= 1; i2++)
            {
                for (int j2 = -1; j2 <= 1; j2++)
                {
                    int x = j + j2;
                    int y = i + i2;
                    if (x >= 0 && x < boardSize && y >= 0 && y < boardSize)
                    {
                        if (i2 != 0 || j2 != 0)
                        {
                            if (cells[y * boardSize + x] == 1)
                            {
                                numNeighbors++;
                            }
                        }
                    }
                }
            }
            neighbors[index] = numNeighbors;
        }
    }
}


__global__ void interop_update_neighbors_kernel(const char* dirty, unsigned char* neighbors, const unsigned int boardSize, const unsigned int ox, const unsigned int oy)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    const unsigned int x = i * sliceSize + ox;
    const unsigned int y = j * sliceSize + oy;

    if (x >= boardSize || y >= boardSize)
        return;

    const unsigned int index = y * boardSize + x;
    const char d = dirty[index];
    if (d == 0)
        return;

    for (int i2 = -1; i2 <= 1; i2++)
    {
        for (int j2 = -1; j2 <= 1; j2++)
        {
            int x2 = x + j2;
            int y2 = y + i2;
            if (x2 >= 0 && x2 < boardSize && y2 >= 0 && y2 < boardSize)
            {
                if (x2 == x && y2 == y)
                {
                    continue;
                }
                neighbors[y2 * boardSize + x2] += d;
            }
        }
    }
}


__global__ void update_all_interop_kernel(const char* dirty, unsigned char* neighbors, const unsigned int boardSize)
{
    for (int y = 0; y < boardSize; y++)
    {
        for (int x = 0; x < boardSize; x++)
        {
            const unsigned int index = y * boardSize + x;
            const char d = dirty[index];
            if (d == 0)
                continue;

            for (int i2 = -1; i2 <= 1; i2++)
            {
                for (int j2 = -1; j2 <= 1; j2++)
                {
                    int x2 = x + j2;
                    int y2 = y + i2;
                    if (x2 >= 0 && x2 < boardSize && y2 >= 0 && y2 < boardSize)
                    {
                        if (x2 == x && y2 == y)
                        {
                            continue;
                        }
                        neighbors[y2 * boardSize + x2] += d;
                    }
                }
            }
        }
    }
}

__global__ void interop_update_cell(unsigned char* cells, unsigned char* cells2, char* dirty, const unsigned char* neighbors, unsigned int boardSize, float* colorsPtr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y >= boardSize)
        return;

    x *= numCellsPerIteration;

    for (int t = 0; t < numCellsPerIteration; t++)
    {
        if (x + t >= boardSize)
            return;

        int index = y * boardSize + x + t;
        int numNeighbors = neighbors[index];
        if (cells[index] == 1)
        {
            if (numNeighbors < 2 || numNeighbors > 3)
            {
                cells2[index] = 0;
                dirty[index] = -1;

                colorsPtr[index * 4] = 0.0f;
                colorsPtr[index * 4 + 1] = 0.0f;
                colorsPtr[index * 4 + 2] = 0.0f;
                colorsPtr[index * 4 + 3] = 0.0f;
            }
        }
        else
        {
            if (numNeighbors == 3)
            {
                cells2[index] = 1;

                colorsPtr[index * 4] = 1.0f;
                colorsPtr[index * 4 + 1] = 1.0f;
                colorsPtr[index * 4 + 2] = 1.0f;
                colorsPtr[index * 4 + 3] = 1.0f;

                dirty[index] = 1;
            }
        }
    }
}


__global__ void interop_merge_cells(unsigned char* cells, unsigned char* cells2, unsigned char* uploadCells, unsigned int boardSize, float* colorsPtr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= boardSize || y >= boardSize)
        return;

    int index = y * boardSize + x;
    if (uploadCells[index] == 1)
    {
        cells[index] = 1;
        cells2[index] = 1;

        colorsPtr[index * 4] = 1.0f;
        colorsPtr[index * 4 + 1] = 1.0f;
        colorsPtr[index * 4 + 2] = 1.0f;
        colorsPtr[index * 4 + 3] = 1.0f;
    }
}

void interop_init(unsigned char* initialCells, const unsigned int boardSize)
{
    interop_boardSize = boardSize;
    hipMalloc((void**)&interop_cells, boardSize * boardSize * sizeof(unsigned char));
    hipMalloc((void**)&interop_cells2, boardSize * boardSize * sizeof(unsigned char));
    hipMalloc((void**)&interop_uploadCells, boardSize * boardSize * sizeof(unsigned char));
    hipMalloc((void**)&interop_neighbors, boardSize * boardSize * sizeof(unsigned char));
    hipMalloc((void**)&interop_dirty, boardSize * boardSize * sizeof(char));

    hipMemcpy(interop_cells, initialCells, boardSize * boardSize * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(interop_cells2, initialCells, boardSize * boardSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    hipMemset(interop_neighbors, 0, boardSize * boardSize * sizeof(unsigned char));
    interop_count_intitial_neighbors<<<1, 1>>>(interop_cells, interop_neighbors, boardSize);
}

void interop_reupload(unsigned char* uploadCells, float* colorsPtr)
{
    hipMemcpy(interop_uploadCells, uploadCells, interop_boardSize * interop_boardSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize(interop_boardSize / 16 + 1, interop_boardSize / 16 + 1);
    interop_merge_cells<<<gridSize, blockSize>>>(interop_cells, interop_cells2, interop_uploadCells, interop_boardSize, colorsPtr);

    hipMemset(interop_neighbors, 0, interop_boardSize * interop_boardSize * sizeof(unsigned char));
    interop_count_intitial_neighbors << <1, 1 >> > (interop_cells, interop_neighbors, interop_boardSize);
}

void interop_update(float* colorsDevicePtr)
{
    hipMemset(interop_dirty, 0, interop_boardSize * interop_boardSize * sizeof(char));

    const int N = 32;

    dim3 blockSize(N, N);
    dim3 gridSize(interop_boardSize / (N* numCellsPerIteration) + 1, interop_boardSize / N + 1);

    interop_update_cell<<<gridSize, blockSize>>>(interop_cells, interop_cells2, interop_dirty, interop_neighbors, interop_boardSize, colorsDevicePtr);

    hipMemcpy(interop_cells, interop_cells2, interop_boardSize * interop_boardSize * sizeof(unsigned char), hipMemcpyDeviceToDevice);

    dim3 nBlockSize(N, N);
    dim3 nGridSize(interop_boardSize / N / sliceSize + 1, interop_boardSize / N / sliceSize + 1);

    for (int y = 0; y < sliceSize; y++)
    {
        for (int x = 0; x < sliceSize; x++)
        {
            interop_update_neighbors_kernel<<<nBlockSize, nGridSize>>>(interop_dirty, interop_neighbors, interop_boardSize, x, y);
        }
    }
}

void interop_destroy()
{
    hipFree(interop_cells);
    hipFree(interop_cells2);
    hipFree(interop_uploadCells);
    hipFree(interop_neighbors);
    hipFree(interop_dirty);
}
